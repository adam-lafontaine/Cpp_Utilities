#include "device.hpp"

#include <hip/hip_runtime.h>
#include <cassert>

#define CUDA_PRINT_ERROR

#ifdef CUDA_PRINT_ERROR

#include <cstdio>

#endif


static void check_error(hipError_t err)
{
    if(err == hipSuccess)
    {
        return;
    }

    #ifdef CUDA_PRINT_ERROR

    printf("\n*** CUDA ERROR ***\n\n");
    printf("%s", hipGetErrorString(err));
    printf("\n\n******************\n\n");

    #endif
}


namespace device
{
    bool malloc(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(!buffer.data);

        hipError_t err = hipMalloc((void**)&(buffer.data), n_bytes);
        check_error(err);

        bool result = err == hipSuccess;

        if(result)
        {
            buffer.capacity = n_bytes;
        }
        
        return result;
    }


    bool free(MemoryBuffer& buffer)
    {
        buffer.capacity = 0;
        buffer.size = 0;

        if(buffer.data)
        {
            hipError_t err = hipFree(buffer.data);
            check_error(err);

            buffer.data = nullptr;

            return err == hipSuccess;
        }

        return true;
    }


    u8* push_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size < buffer.capacity);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size < buffer.capacity;

        auto bytes_available = (buffer.capacity - buffer.size) >= n_bytes;
        assert(bytes_available);

        if(!is_valid || !bytes_available)
        {
            return nullptr;
        }

        auto data = buffer.data + buffer.size;

        buffer.size += n_bytes;

        return data;
    }


    bool pop_bytes(MemoryBuffer& buffer, size_t n_bytes)
    {
        assert(buffer.data);
        assert(buffer.capacity);
        assert(buffer.size <= buffer.capacity);
        assert(n_bytes <= buffer.capacity);
        assert(n_bytes <= buffer.size);

        auto is_valid = 
            buffer.data &&
            buffer.capacity &&
            buffer.size <= buffer.capacity &&
            n_bytes <= buffer.capacity &&
            n_bytes <= buffer.size;

        if(is_valid)
        {
            buffer.size -= n_bytes;
            return true;
        }

        return false;
    }
}